
#include <hip/hip_runtime.h>
__global__ void sumCommSingleBlock(const int *a, int *out) {

    static const int arraySize = 10000;
    static const int blockSize = 1024;

    int idx = threadIdx.x;
    int sum = 0;
    for (int i = idx; i < arraySize; i += blockSize)
        sum += a[i];
    __shared__ int r[blockSize];
    r[idx] = sum;
    __syncthreads();
    for (int size = blockSize/2; size>0; size/=2) { //uniform
        if (idx<size)
            r[idx] += r[idx+size];
        __syncthreads();
    }
    if (idx == 0)
        *out = r[0];
}