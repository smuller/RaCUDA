#include "hip/hip_runtime.h"
__global__ addSubArray1 ( int *A,  int *B,  int w,  int h, int bsx) {

    signed long int size_A1;
    signed long int lower_bound_A1 = 0;
    signed long int upper_bound_A1;
    upper_bound_A1 = w + -1;
    lower_bound_A1 = 0;
    size_A1 = upper_bound_A1 - lower_bound_A1;
    __shared__ int A1[size_A1];
    //for (signed long int __itertemp =  0; __itertemp <= size_A1; __itertemp = __itertemp
    //                                                                + blockSize.x) {
    
        A1[(blockIdx.x * blockDim.x) + threadIdx.x] = A[(blockIdx.x * blockDim.x) + threadIdx.x + lower_bound_A1]; 
    //}
    signed long int size_B1;
    signed long int lower_bound_B1;
    signed long int upper_bound_B1;
    upper_bound_B1 = (w * 2)
                     + (((w * threadIdx.x) * 2)
                        + (((w * (blockIdx.x * blockDim.x)) * 2) + -1));
    lower_bound_B1 = w
                     + (((w * threadIdx.x) * 2)
                        + ((w * (blockIdx.x * blockDim.x)) * 2));
    size_B1 = upper_bound_B1 - lower_bound_B1;
    __shared__ int B1[size_B1];
    //for (signed long int __itertemp =  0; __itertemp <= w - 1; __itertemp = __itertemp
    //                                                                + bsx) {
    
        B1[(blockIdx.x * blockDim.x) + threadIdx.x] = B[(blockIdx.x * blockDim.x) + threadIdx.x + lower_bound_B1]; 
    //}
    signed long int size_B2;
    signed long int lower_bound_B2;
    signed long int upper_bound_B2;
    upper_bound_B2 = w
                     + (((w * threadIdx.x) * 2)
                        + (((w * (blockIdx.x * blockDim.x)) * 2) + -1));
    lower_bound_B2 = ((w * threadIdx.x) * 2)
                     + ((w * (blockIdx.x * blockDim.x)) * 2);
    size_B2 = upper_bound_B2 - lower_bound_B2;
    __shared__ int B2[size_B2];
    //for (signed long int __itertemp =  0; __itertemp <= size_B2; __itertemp = __itertemp
    //                                                                + blockSize.x) {
    
        B2[(blockIdx.x * blockDim.x) + threadIdx.x] = B[(blockIdx.x * blockDim.x) + threadIdx.x + lower_bound_B2]; 
    //}
    for (int i = 0; i < w; i = i + 1) {
    
        int j;
        j = (blockIdx.x * blockDim.x) + threadIdx.x;
        B2[(((2 * j) * w) + i) - lower_bound_B2] = B2
                                                   [(((2 * j) * w) + i)
                                                    - lower_bound_B2]
                                                   + A[i];
        B1[((((2 * j) + 1) * w) + i) - lower_bound_B1] = B1
                                                         [((((2 * j) + 1) * w)
                                                           + i)
                                                          - lower_bound_B1]
                                                         - A
                                                           [i]; 
    }
    //for (signed long int __itertemp =  0; __itertemp <= size_A1; __itertemp = __itertemp
    //                                                                + blockSize.x) {
    
        A[(blockIdx.x * blockDim.x) + threadIdx.x + lower_bound_A1] = A1[(blockIdx.x * blockDim.x) + threadIdx.x]; 
    //}
    //for (signed long int __itertemp =  0; __itertemp <= size_B1; __itertemp = __itertemp
    //                                                                + blockSize.x) {
    
        B[(blockIdx.x * blockDim.x) + threadIdx.x + lower_bound_B1] = B1[(blockIdx.x * blockDim.x) + threadIdx.x]; 
   // }
    //for (signed long int __itertemp =  0; __itertemp <= size_B2; __itertemp = __itertemp
    //                                                                + blockSize.x) {
    
        B[(blockIdx.x * blockDim.x) + threadIdx.x + lower_bound_B2] = B2[(blockIdx.x * blockDim.x) + threadIdx.x]; 
    //} 
}

