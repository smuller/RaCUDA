#include "hip/hip_runtime.h"
/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

__global__ void reduce2(int *g_idata, int *g_odata, unsigned int n)
{
  __shared__ int sdata[n];

  // load shared mem
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

  if (i < n) {
    sdata[tid] = g_idata[i];
  }


  // do reduction in shared mem
  for (unsigned int s=blockDim.x/2; s>0; s/=2)
    {
      if (tid < s)
        {
          sdata[tid] += sdata[tid + s];
        }

    }

  // write result for this block to global mem
  if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}