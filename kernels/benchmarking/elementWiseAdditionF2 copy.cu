#include <iostream>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>


// Use a constant for TILE_DIM, e.g., 32
#define TILE_DIM 32

__global__ void elementWiseAddition ( float *a,  float *b,  float *c,  int N) {

    signed long int lower_bound_a1;
    lower_bound_a1 = 0;
    __shared__ float a1[TILE_DIM];
    a1[threadIdx.x] = a[((blockIdx.x * blockDim.x) + threadIdx.x) + lower_bound_a1];
    signed long int lower_bound_b1;
    lower_bound_b1 = 0;
    __shared__ float b1[TILE_DIM];
    b1[threadIdx.x] = b[((blockIdx.x * blockDim.x) + threadIdx.x) + lower_bound_b1];
    signed long int lower_bound_c1;
    lower_bound_c1 = 0;
    __shared__ float c1[TILE_DIM];
    c1[threadIdx.x] = c[((blockIdx.x * blockDim.x) + threadIdx.x) + lower_bound_c1];
    int i;
    for (i = 0; i < N; i = i + 1) {
    
        int j;
        for (j = 0; j < N; j = j + 1) {
        
            c1[threadIdx.x - lower_bound_c1] = a1[threadIdx.x - lower_bound_a1]+ b1[threadIdx.x - lower_bound_b1]; 
        } 
    }
    a[((blockIdx.x* blockDim.x) + threadIdx.x) + lower_bound_a1] = a1[threadIdx.x];
    b[((blockIdx.x* blockDim.x) + threadIdx.x) + lower_bound_b1] = b1[threadIdx.x];
    c[((blockIdx.x* blockDim.x) + threadIdx.x) + lower_bound_c1] = c1[threadIdx.x]; 
}


void checkCudaError(hipError_t err, const char *msg)
{
  if (err != hipSuccess)
  {
    std::cerr << "CUDA error: " << msg << ": " << hipGetErrorString(err) << std::endl;
    exit(EXIT_FAILURE);
  }
}

int main()
{
  int N = 1 << 20; // Array size (e.g., 2^20 elements)

  // Allocate and initialize host vectors a, b, and c
  std::vector<float> a(N, 1.0f);
  std::vector<float> b(N, 2.0f);
  std::vector<float> c(N, 0.0f);

  // Allocate device memory for a, b, and c
  float *d_a;
  float *d_b;
  float *d_c;
  checkCudaError(hipMalloc(&d_a, N * sizeof(float)), "allocating d_a");
  checkCudaError(hipMalloc(&d_b, N * sizeof(float)), "allocating d_b");
  checkCudaError(hipMalloc(&d_c, N * sizeof(float)), "allocating d_c");

  // Copy host memory to device
  checkCudaError(hipMemcpy(d_a, a.data(), N * sizeof(float), hipMemcpyHostToDevice), "copying a to d_a");
  checkCudaError(hipMemcpy(d_b, b.data(), N * sizeof(float), hipMemcpyHostToDevice), "copying b to d_b");

  // Configure the kernel launch parameters
  int blockSize = TILE_DIM;
  int gridSize = (N + blockSize - 1) / blockSize;

  // Measure kernel execution time
  hipEvent_t start, stop;
  checkCudaError(hipEventCreate(&start), "creating start event");
  checkCudaError(hipEventCreate(&stop), "creating stop event");

  // Launch the kernel
  checkCudaError(hipEventRecord(start), "recording start event");
  elementWiseAddition<<<gridSize, blockSize>>>(d_a, d_b, d_c, N);
  checkCudaError(hipEventRecord(stop), "recording stop event");

  // Synchronize and check for errors
  checkCudaError(hipEventSynchronize(stop), "synchronizing on stop event");
  checkCudaError(hipGetLastError(), "launching elementWiseAddition kernel");

  // Calculate elapsed time
  float elapsedTime;
  checkCudaError(hipEventElapsedTime(&elapsedTime, start, stop), "calculating elapsed time");

  // Copy the result from device to host
  checkCudaError(hipMemcpy(c.data(), d_c, N * sizeof(float), hipMemcpyDeviceToHost), "copying d_c to c");

  // Verify the result
  bool success = true;
  for (int i = 0; i < N; i++)
  {
    if (c[i] != a[i] + b[i])
    {
      success = false;
      break;
    }
  }

  if (success)
  {
    std::cout << "Element-wise addition succeeded!" << std::endl;
    std::cout << "Elapsed time: " << elapsedTime << " ms" << std::endl;
  }
  else
  {
    std::cerr << "Element-wise addition failed!" << std::endl;
  }

  // Clean up resources
  checkCudaError(hipEventDestroy(start), "destroying start event");
  checkCudaError(hipEventDestroy(stop), "destroying stop event");
  checkCudaError(hipFree(d_a), "freeing d_a");
  checkCudaError(hipFree(d_b), "freeing d_b");
  checkCudaError(hipFree(d_c), "freeing d_c");

  return (success ? EXIT_SUCCESS : EXIT_FAILURE);
}