
#include <hip/hip_runtime.h>
__global__ void addSubArray1 (int *A, int *B, int w, int h) {

  //if (h > 4) {
    for (int i = 0; i < w; i++) {
      int j = blockIdx.x * blockDim.x + threadIdx.x;
      B[2 * j * w + i] += A[i];
      B[(2 * j + 1) * w + i] -= A[i];
    }
  //} else {
    for (int i = 0; i < 5; i++) {
      //int k = 2 + 1 - 3 * 5 + w;
      //int j = 9 - 3 + k * 2;
      //A[i] = 1;
      //B[i] = 1;
      B[i] += A[i];
      B[i] -= A[i];
    }
  //}
  //B[w] = 0;
}
