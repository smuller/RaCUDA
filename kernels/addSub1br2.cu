
#include <hip/hip_runtime.h>
__global__ void addSubArray1 (int *A, int *B, int w, int h) {
  for (int i = 0; i < w; i++) {
    if (i == 3) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
      B[2 * j * w + i] += A[i];
      B[(2 * j + 1) * w + i] -= A[i];
    } else {
      int j = blockIdx.x * blockDim.x + threadIdx.x;
      B[2 * j * w + i] += A[i];
      B[(2 * j + 1) * w + i] -= A[i];
    }
    
  }
}